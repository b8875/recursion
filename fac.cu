#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 16

__global__ void factd(int64_t *array, int n, int index){

	if (n > 0){
		factd<<<1, 1>>>(array, n - 1, index);
		array[index] +=  n;	
	}
}

__global__ void factorial(int64_t *array){
	int id = threadIdx.x + blockDim.x * blockIdx.x;
	factd<<<1, 1>>>(array, id, id);

}


int main()
{
	int64_t* host = new int64_t[N];
	int64_t* device;
	hipMalloc( (void**)&device, N * sizeof( int64_t ) );
	size_t pValue;
        hipDeviceSetLimit(hipLimitStackSize, 8192);
        hipDeviceGetLimit(&pValue, hipLimitStackSize);

	for( unsigned int i = 1; i < N; ++i )
	{
		host[i] = 0;
	}

	hipMemcpy( device, host, N * sizeof( int64_t ), hipMemcpyHostToDevice );
	
	hipEvent_t     start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start, 0);

	factorial<<< 1, N >>>( device );
	
	hipEventRecord(stop, 0);
        hipEventSynchronize(stop);

        float elapsedTime;
        hipEventElapsedTime(&elapsedTime, start, stop);
        hipEventDestroy(start);
        hipEventDestroy(stop);

	printf("%f ms\n",elapsedTime);
	hipMemcpy( host, device, N * sizeof( int64_t ), hipMemcpyDeviceToHost );
#if 0
	for (int i = 0; i < N; i++){
		printf("%ld, %d\n", host[i], i);
	}
#endif 
	hipFree( device );
	delete[] host;
}
